#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../cuda_utils.cuh"

/*
  Function: grouping features of neighbors (forward)
  Args:
    b   : batch size
    c   : #channles of features
    n   : number of points in point clouds
    m   : number of query centers
    u   : maximum number of neighbors
    features: points' features, FloatTensor[b, c, n]
    indices : neighbor indices in points, IntTensor[b, m, u]
    out     : gathered features, FloatTensor[b, c, m, u]
*/
__global__ void grouping_kernel(int b, int c, int n, int m, int u,
                                const float *__restrict__ features,
                                const int *__restrict__ indices,
                                float *__restrict__ out) {
  // locate current batch
  int batch_index = blockIdx.x; // why not const ?
  features += batch_index * n * c;
  indices += batch_index * m * u;
  out += batch_index * m * u * c;

  // dim2 parallel, first each center points (m), then each feature channel (c)
  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * m; i += stride) {
    const int l = i / m; // l is the current feature channel 
    const int j = i % m; // j is the current center point 
    // k is the current neighbour
    for (int k = 0; k < u; ++k) {
      // out[i * u + k]
      out[(l * m + j) * u + k] = features[l * n + indices[j * u + k]];
    }
  }
}

void grouping(int b, int c, int n, int m, int u, const float *features,
              const int *indices, float *out) {
  // launch batch_size blocks, divided into num_center_points * num_channels threads
  // because output is [b, c, m, u], we parallel it as [b, c*m, u]
  grouping_kernel<<<b, optimal_block_config(m, c), 0, at::cuda::getCurrentCUDAStream()>>>(
    b, c, n, m, u, features, indices, out
  );
  CUDA_CHECK_ERRORS();
}

/*
  Function: grouping features of neighbors (backward)
  Args:
    b   : batch size
    c   : #channles of features
    n   : number of points in point clouds
    m   : number of query centers
    u   : maximum number of neighbors
    grad_y : grad of gathered features, FloatTensor[b, c, m, u]
    indices : neighbor indices in points, IntTensor[b, m, u]
    grad_x: grad of points' features, FloatTensor[b, c, n]
*/
__global__ void grouping_grad_kernel(int b, int c, int n, int m, int u,
                                     const float *__restrict__ grad_y,
                                     const int *__restrict__ indices,
                                     float *__restrict__ grad_x) {

  int batch_index = blockIdx.x; 
  grad_y += batch_index * m * u * c;
  indices += batch_index * m * u;
  grad_x += batch_index * n * c;
  
  // dim2 parallel
  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * m; i += stride) {
    const int l = i / m;
    const int j = i % m;
    // atomic because multiple threads may work on the same address
    for (int k = 0; k < u; ++k) {
      atomicAdd(grad_x + l * n + indices[j * u + k], grad_y[(l * m + j) * u + k]);
    }
  }
}

void grouping_grad(int b, int c, int n, int m, int u, const float *grad_y, const int *indices, float *grad_x) {
  grouping_grad_kernel<<<b, optimal_block_config(m, c), 0, at::cuda::getCurrentCUDAStream()>>>(
    b, c, n, m, u, grad_y, indices, grad_x
  );
  CUDA_CHECK_ERRORS();
}
